#include <hip/hip_runtime.h>

#include <stdio.h>


int main(int argc, char **argv) {
  double *buf_d = NULL;
  fprintf(stderr, "Allocating...\n");
  hipMalloc((void **) &buf_d, sizeof(double) * 1024);
  fprintf(stderr, "Allocating DONE.\n");
  return 0;
}
